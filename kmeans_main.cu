
#include <stdio.h>
#include <stdlib.h>
#include <string.h>   
#include <sys/types.h> 
#include <sys/stat.h>
#include <random>
#include <vector>

#include "kmeans.h"
#include "file_utils.h"
#include "Read_MNIST.h"

#define MAX_MEAN 1000

int main(int argc, char **argv) {

	/*
	CHECK(hipDeviceReset());
	int deviceNum = 0;
	hipDeviceProp_t deviceProp;
	
	hipGetDeviceProperties(&deviceProp, deviceNum);
	printf("[INFO] Device Number: %d\n", deviceNum);
	*/
	km_float** data;
	km_float** centroids;
	/*
#ifdef LOAD_MNIST
	printf("this shouldn't be printed.\n");
    init_data mnistdata;
    readMNISTFloat(&data, "/path/to/labels", "/path/to/images");
    int n = mnistdata.numSamples;
    int k = mnistdata.classes;
    int d = mnistdata.dim;
    data = mnistdata.data;
    int* true_labels;
    true_labels = mnist.labels;
    kmfloat threshold = mnist.threshold;
    int* pred_labels; 
    pred_labels = (int *)malloc(n * sizeof(int));

#else
*/
    printf("Not using any dataset, generating random data specified in parameters!\n");
    // Load Parameters
	parameters params;

	km_float threshold = params.threshold;
	int n = params.numSamples;
	int k = params.classes;
	int d = params.dim;
   
	int loop_iterations;

	int* pred_labels;
	pred_labels = (int *)malloc(n * sizeof(int));
    assert(pred_labels != NULL);
	// Allocate Memory
	printf("[INFO]: Allocating Memory\n");
	try {
		malloc2D(data, n, d, km_float);
	}
	catch (std::bad_alloc const &e) {
		free(data[0]);
		free(data);
	}

	// Generate Random Data of varying mean, with stddev 2.0
	printf("[INFO]: Generating Random Values\n");
	std::default_random_engine generator;
	//km_float *means = new km_float(k);
    //km_float *means_y = new km_float(k);
	km_float *means;
	km_float *means_y;
	means = (km_float*)malloc(sizeof(km_float)*k);
	means_y = (km_float*)malloc(sizeof(km_float)*k);;
	int count = 0;
    /*
	for (int i = 0; i < k; i++) {
		means[i] = (km_float)count;
		count += 5;
	}
    */
    for(int i = 0; i < k; i++) {
        means[i] = rand() % MAX_MEAN + 1;
        means_y[i] = rand() % MAX_MEAN + 1;
    }
	int pointsPerLabel = n / k;
	//km_float mean;
    //km_float mean_y;
	for (int i = 0; i < k; i++) {
        // Sample from random distribution for varying X and Y means
		//mean_x = means[i];
        //mean_y = means_y[i];

		std::normal_distribution<km_float> distribution_x(means[i], 150.0);
        std::normal_distribution<km_float> distribution_y(means_y[i], 150.0);
		for (int j = 0; j < pointsPerLabel; j++) {
			for (int z = 0; z < d; z++) {
                km_float num;
                if(z == 0) { 
                    num = distribution_x(generator);
                } else {
                    num = distribution_y(generator);
                }
				data[i * pointsPerLabel + j][z] = num;
			}

		}
	}
	if (data == NULL) {
        exit(1);
    }
//#endif
	const char* file_name = "kmeans1.bin";
	FILE *f = fopen(file_name, "wb");
	int saved = log_points(data, f, 1, k, n, d);
	//int saved = save_points(data, file_name, 1, k, n, d);
	if (saved == 0) {
		printf("[FILE] %d data points saved\n\n", n);
	}

	centroids = cu_kmeans(data,
		threshold,
		pred_labels,
		&loop_iterations,
		d,
		n,
		k,
		f);
	/*
	for (int a = 0; a < k; a++) {
		printf("centroids %d: ", a);
		for (int b = 0; b < d; b++) {
			printf("%f ", centroids[a][b]);
		}
		printf("\n\n");
	}
	*/
	fclose(f);
    free(data[0]);
    free(data);
    free(pred_labels);
    free(centroids[0]);
    free(centroids);
    free(means);
    free(means_y);

    return(0);
}

