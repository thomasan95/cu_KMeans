
#include <stdio.h>
#include <stdlib.h>
#include <string.h>     /* strtok() */
#include <sys/types.h>  /* open() */
#include <sys/stat.h>
#include <random>
#include <vector>


int      _debug;
#include "kmeans.h"


int main(int argc, char **argv) {

	km_float ** data;
	km_float ** centroids;

    _debug = 1;
	// Load Parameters
	parameters params;

	km_float threshold = params.threshold;
	int n = params.numSamples;
	int k = params.classes;
	int d = params.dim;
   
	int loop_iterations;

	int* labels;
	labels = (int *)malloc(sizeof(int) * n);

	// Allocate Memory
	printf("[INFO]: Allocating Memory\n");
	try {
		malloc2D(data, n, d, km_float);
	}
	catch (std::bad_alloc const &e) {
		free(data[0]);
		free(data);
	}

	// Generate Random Data of varying mean, with stddev 2.0
	printf("[INFO]: Generating Random Values\n");
	std::default_random_engine generator;
	km_float *means = new km_float(k);
	int count = 0;
	for (int i = 0; i < k; i++) {
		means[i] = (km_float)count;
		count += 5;
	}
	int pointsPerLabel = n / k;
	km_float mean = 0.0;

	for (int i = 0; i < k; i++) {
		mean = means[i];
		std::normal_distribution<km_float> distribution(mean, 2.0);
		for (int j = 0; j < pointsPerLabel; j++) {
			for (int z = 0; z < d; z++) {
				km_float num = distribution(generator);
				data[i * pointsPerLabel + j][z] = num;
			}

		}
	}    
	if (data == NULL) exit(1);

	labels = (int*) malloc(n * sizeof(int));
    assert(labels != NULL);

    centroids = cu_kmeans(data, d, n, k, threshold, labels, &loop_iterations);

	for (int a = 0; a < k; a++) {
		printf("centroids %d: ", a);
		for (int b = 0; b < d; b++) {
			printf("%f ", centroids[a][b]);
		}
		printf("\n\n");
	}



    free(data[0]);
    free(data);
    free(labels);
    free(centroids[0]);
    free(centroids);


    return(0);
}

