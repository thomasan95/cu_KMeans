#include "hip/hip_runtime.h"
// #include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <cstring>    
#include <sys/types.h> 
#include <sys/stat.h>

#include "file_utils.h"

#define MAX_CHAR_PER_LINE 128
#define _debug 1


/** @brief reads file from specified path
*	@param
*		char* path: path to read file from
*		int isBinary: Whether file in binary format or not
*		int *numSamples: write number of samples to numSamples
*		int *dim: write dimension to dim
*	@return
*		km_float** data stored in 2D array of [numSamples][dim]
*/
km_float** read_file(char* path, int isBinary, int* numSamples, int* dim) {
	km_float **data;
	int count;
	int len;

	if (isBinary) {
		FILE* fptr = fopen(path, "rb");

		if (fptr == NULL) {
			printf("Error reading file %s\n", path);
			exit(0);
		}
		count = fread(numSamples, sizeof(int), 1, fptr);
		assert(count == 1);
		count = fread(dim, sizeof(int), 1, fptr);
		assert(count == 1);
		if (_debug) {
			printf("File %s numSamples  = %d\n", path, *numSamples);
			printf("File %s dims        = %d\n", path, *dim);
		}

		data = (km_float**)malloc((*numSamples) * sizeof(km_float*));
		assert(data != NULL);
		data[0] = (km_float*)malloc((*numSamples) * (*dim) * sizeof(km_float));
		assert(data[0] != NULL);
		for (int i = 1; i < (*numSamples); i++) {
			// Set pointers to each data point
			data[i] = data[i - 1] + (*dim);
		}
		count = fread(data[0], sizeof(km_float), (*dim)*(*numSamples), fptr);
		assert(count == (*dim)*(*numSamples));

		fclose(fptr);
	}
	else {
		FILE *fptr = fopen(path, "r");
		char *line, *ret;
		int curLen;

		if (fptr == NULL) {
			printf("Error reading file %s\n", path);
			return NULL;
		}
		curLen = MAX_CHAR_PER_LINE;
		line = (char*)malloc(curLen);
		assert(line != NULL);
		(*numSamples) = 0;
		while (fgets(line, curLen, fptr) != NULL) {
			while (strlen(line) == curLen - 1) {
				// Not complete line read
				len = strlen(line);
				fseek(fptr, -len, SEEK_CUR);

				curLen += MAX_CHAR_PER_LINE;
				// Reallocate to larger memory
				line = (char*)realloc(line, curLen);
				assert(line != NULL);

				ret = fgets(line, curLen, fptr);
				assert(ret != NULL);
			}
			if (strtok(line, "\t\n") != 0) {
				(*numSamples)++;
			}
		}
		rewind(fptr);
		if (_debug) {
			printf("curLen = %d\n", curLen);
		}
		(*dim) = 0;
		while (fgets(line, curLen, fptr) != NULL) {
			if (strtok(line, "\t\n") != 0) {
				/* ignore the id (first coordinate): dim = 1; */
				while (strtok(NULL, " ,\t\n") != NULL) (*numSamples)++;
				break;
			}
		}
		rewind(fptr);
		if (_debug) {
			printf("File %s numSamples = %d\n", path, *numSamples);
			printf("File %s dim        = %d\n", path, *dim);
		}

		data = (km_float**)malloc((*numSamples) * sizeof(km_float*));
		assert(data != NULL);
		// Set [0]th pointer to start of data
		data[0] = (km_float*)malloc((*numSamples) * (*dim) * sizeof(km_float));
		assert(data[0] != NULL);
		for (int i = 1; i < (*numSamples); i++) {
			// Set subsequent pointer to next data point
			data[i] = data[i - 1] + (*dim);
		}
		int i = 0;
		while (fgets(line, curLen, fptr) != NULL) {
			if (strtok(line, " \t\n") == NULL) continue;
			for (int j = 0; j < (*dim); j++) {
				data[i][j] = atof(strtok(NULL, " ,\t\n"));
			}
			i++;
		}
		fclose(fptr);
		free(line);
	}
	return data;
}


/** @brief Function for saving the model
*	@param
*		kmeans_model const* model: model to save
*		char const *path: save path
*	@return 0 if success 1 if failed
*/
int log_centroids(km_float** centroids, FILE* f, int isBinary, int k, int d) {

	clock_t start;
	start = clock();

	// Write in Binary
	if (isBinary) {
		for (int i = 0; i < k; i++) {
			for (int j = 0; j < d; j++) {
				fwrite(&centroids[j][i], sizeof(km_float), 1, f);
			}
		}
	}
	else {

		for (int i = 0; i < k; i++) {
			fprintf(f, "%d ", i);
			for (int j = 0; j < d; j++) {
				fprintf(f, "%f ", centroids[i][j]);
			}
			fprintf(f, "\n");
		}

	}
	//printf("Time Elapsed: %.8lfs\n", (clock() - start) / (double)CLOCKS_PER_SEC);
	return 0;
}

/** @brief Function for saving the cluster of the current point
*	@param
*		int* currCluster: current labels of the points
*		char const *path: path to save the label to
*		int isBinary: Whether file in binary format or not
*		int n: number of points to save
*	@return 0 if success 1 if failed
*/
int log_labels(int* labels, FILE *f, int isBinary, int n) {
	//printf("\n========== Saving Labels ==========\n");

	clock_t start;
	start = clock();

	char command[1024];
	//sprintf(command, "del %s", path);
	//int sys_ret = system(command);

	// Write in Binary
	if (isBinary) {
		//data points and centroids already written to file
		fwrite(labels, sizeof(int), n, f);
	}
	else {

		for (int i = 0; i < n; i++) {
			fprintf(f, "%d ", i);
			fprintf(f, "%i\n", labels[i]);
		}
	}
	//printf("Time Elapsed: %.8lfs\n", (clock() - start) / (double)CLOCKS_PER_SEC);
	return 0;
}


/** @brief Function for saving the random generated points
*	@param
*		km_float** data: data points to save
*		char const *path: path to save the points to
*		int isBinary: Whether file in binary format or not
*		int k: number of classes
*		int n: number of points to save
*		int d: dimension of the point
*	@return 0 if success 1 if failed
*/
int log_points(km_float** data, FILE*f, int isBinary, int k, int n, int d) {
	printf("\n==========Saving Points ==========\n");

	clock_t start;
	start = clock();

	//char command[1024];
	//sprintf(command, "del %s", path);
	//int sys_ret = system(command);

	// Write in Binary
	if (isBinary) {

		fwrite(&k, sizeof(int), 1, f);
		fwrite(&n, sizeof(int), 1, f);
		fwrite(&d, sizeof(int), 1, f);

		for (int i = 0; i < n; i++) {
			for (int j = 0; j < d; j++) {
				fwrite(&data[i][j], sizeof(km_float), 1, f);
			}
		}
	}
	else {

		for (int i = 0; i < n; i++) {
			fprintf(f, "%d ", i);
			for (int j = 0; j < d; j++) {
				fprintf(f, "%f ", data[i][j]);
			}
			fprintf(f, "\n");
		}

	}
	printf("Time Elapsed: %.8lfs\n", (clock() - start) / (double)CLOCKS_PER_SEC);
	return 0;
}